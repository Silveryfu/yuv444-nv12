#include "hip/hip_runtime.h"
#include "utility.h"
#define BLOCK_WIDTH 32
#ifndef __HIPCC__  
    #define __HIPCC__
#endif

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//  Subsampling from YUV444 to NV12
//	YUV 4:2:0 image with a plane of 8 bit Y samples followed 
//	by an interleaved U/V plane containing 8 bit 2x2 subsampled 
//	colour difference samples.
// 						Horizontal	Vertical
//		Y	   Sample Period	1	1
//		U (Cb) Sample Period	2	2
//		V (Cr) Sample Period	2	2

__global__ void yuv2nv(unsigned char * y_in,unsigned char * u_in,unsigned char * v_in, 
	unsigned char * y_out,unsigned char * u_out,unsigned char * v_out, int img_width, int img_height){
		__shared__ unsigned char sample[BLOCK_WIDTH][BLOCK_WIDTH][2];

		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int row = blockIdx.y * BLOCK_WIDTH + ty;
		int col = blockIdx.x * BLOCK_WIDTH + tx;

		unsigned int index = row * img_width + col;

		//@@ TODO: need a better way to dispatch the memory access!
		if(ty % 2 == 0 && tx % 2 == 0) {
			if(row < img_height && col < img_width) {
				sample[ty][tx][0] = u_in[index];
				sample[ty][tx][1] = v_in[index];
				sample[ty+1][tx+1][0] = u_in[index];
				sample[ty+1][tx+1][1] = v_in[index];
				sample[ty+1][tx][0] = u_in[index];
				sample[ty+1][tx][1] = v_in[index];
				sample[ty][tx+1][0] = u_in[index];
				sample[ty][tx+1][1] = v_in[index];
			} 
		}

		__syncthreads();

		if(row < img_height && col < img_width) {
			y_out[index] = y_in[index];
			u_out[index] = sample[ty][tx][0];
			v_out[index] = sample[ty][tx][1];
		}
}

//@@ CUDA kernel
__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

//Convert RGB to YUV, all components in [0, 255]
__global__ void rgb2yuv_cuda(unsigned char * img_r,unsigned char * img_g,unsigned char * img_b, 
	unsigned char * img_y,unsigned char * img_u,unsigned char * img_v, int imgSize)
{
    
	int gid = threadIdx.x+blockIdx.x*blockDim.x;

    if(gid < imgSize)
	{
        img_y[gid] = (unsigned char)( 0.299*img_r[gid] + 0.587*img_g[gid] +  0.114*img_b[gid]);
        img_u[gid] = (unsigned char)(-0.169*img_r[gid] - 0.331*img_g[gid] +  0.499*img_b[gid] + 128);
        img_v[gid] = (unsigned char)( 0.499*img_r[gid] - 0.418*img_g[gid] - 0.0813*img_b[gid] + 128);
    }
}

int main()
{
	clock_t start, end;

	unsigned char* host_img_y, *host_img_u, *host_img_v;
	unsigned char* device_img_y_in, *device_img_u_in, *device_img_v_in;
	unsigned char* device_img_y_out, *device_img_u_out, *device_img_v_out;

	PPM_IMG img_in;
	YUV_IMG img_yuv;

	const int arraySize = 5;
	const int a[arraySize] = { 1, 2, 3, 4, 5 };
	const int b[arraySize] = { 10, 20, 30, 40, 50 };
	int c[arraySize] = { 0 };

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	queryDevice();

	img_in = read_ppm("test1.ppm");
	img_yuv = rgb2yuv(img_in);

	// Begin of conversion and subsampling
	start = clock();

	host_img_y = img_yuv.img_y;
	host_img_u = img_yuv.img_u;
	host_img_v = img_yuv.img_v;

	printf("\nTEST: %d\n", host_img_y[10240]);
	printf("TEST: %d\n", host_img_u[10240]);
	printf("TEST: %d\n", host_img_v[10240]);


	myCudaCheck(hipMalloc((void **) &device_img_y_in, img_yuv.h * img_yuv.w * sizeof(unsigned char)));
	myCudaCheck(hipMalloc((void **) &device_img_u_in, img_yuv.h * img_yuv.w * sizeof(unsigned char)));
	myCudaCheck(hipMalloc((void **) &device_img_v_in, img_yuv.h * img_yuv.w * sizeof(unsigned char)));

	myCudaCheck(hipMalloc((void **) &device_img_y_out, img_yuv.h * img_yuv.w * sizeof(unsigned char)));
	myCudaCheck(hipMalloc((void **) &device_img_u_out, img_yuv.h * img_yuv.w * sizeof(unsigned char)));
	myCudaCheck(hipMalloc((void **) &device_img_v_out, img_yuv.h * img_yuv.w * sizeof(unsigned char)));

	myCudaCheck(hipMemcpy(device_img_y_in, host_img_y, img_yuv.h * img_yuv.w * sizeof(unsigned char), hipMemcpyHostToDevice));
	myCudaCheck(hipMemcpy(device_img_u_in, host_img_u, img_yuv.h * img_yuv.w * sizeof(unsigned char), hipMemcpyHostToDevice));
	myCudaCheck(hipMemcpy(device_img_v_in, host_img_v, img_yuv.h * img_yuv.w * sizeof(unsigned char), hipMemcpyHostToDevice));

	dim3 dimGrid((img_yuv.w - 1)/BLOCK_WIDTH + 1, (img_yuv.w - 1)/BLOCK_WIDTH + 1, 1);
	dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH, 1);

	yuv2nv<<<dimGrid, dimBlock>>>(device_img_y_in, device_img_u_in, device_img_v_in, 
								  device_img_y_out, device_img_u_out, device_img_v_out, 
								  img_yuv.w, img_yuv.h);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("Subsampling kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	hipDeviceSynchronize();

	myCudaCheck(hipMemcpy(host_img_y, device_img_y_out, img_yuv.h * img_yuv.w * sizeof(unsigned char), hipMemcpyDeviceToHost));
	myCudaCheck(hipMemcpy(host_img_u, device_img_u_out, img_yuv.h * img_yuv.w * sizeof(unsigned char), hipMemcpyDeviceToHost));
	myCudaCheck(hipMemcpy(host_img_v, device_img_v_out, img_yuv.h * img_yuv.w * sizeof(unsigned char), hipMemcpyDeviceToHost));
	end = clock();
	printf("\nTime taken is: %d seconds %d milliseconds.\n", (end - start)/(CLOCKS_PER_SEC), (end - start)*1000/(CLOCKS_PER_SEC)%1000);
	printf("\nRaw time: %ld\n", end - start);
	// End of conversion and subsampling


	printf("\nTEST: %d\n", img_yuv.img_y[10240]);
	printf("TEST: %d\n", img_yuv.img_u[10240]);
	printf("TEST: %d\n", img_yuv.img_v[10240]);


	img_in = yuv2rgb(img_yuv);
	write_ppm(img_in, "test_out.ppm");

	hipFree(device_img_y_in);
	hipFree(device_img_u_in);
	hipFree(device_img_v_in);
	hipFree(device_img_y_out);
	hipFree(device_img_u_out);
	hipFree(device_img_v_out);

	free_ppm(img_in);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	getchar();
	getchar();

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
